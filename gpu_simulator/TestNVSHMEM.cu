#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#include "hip/hip_fp16.h"
#include "cuda_helper.h"
#include "mpi.h"
#include <nvshmem.h>
#include <nvshmemx.h>

#define PKTSIZE 8
#define NELEM 1024

__global__ void ring_bcast(half *data, size_t nelem, int root, uint64_t *psync) {
    //Bidirectional algorithm. First CTA sends rightwards and second CTA sends in opposite direction. 
    int direction = blockIdx.x % 2;
    size_t offset = direction * (nelem/2);

    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = direction == 0 ?
               (mype + 1) % npes : (mype + npes - 1) % npes;

    size_t npackets = (nelem/2 + PKTSIZE-1) / PKTSIZE;

    uint64_t *mysync = &psync[direction];
    *mysync = 0;
    
    if (direction == 0 && mype == (root + npes - 1) % npes) return;

    if (direction == 1 && mype == (root + 1) % npes) return;

    for (int idx=0; idx < npackets; idx++){
        half* pos = data + offset + idx*PKTSIZE;
        int elemcount = idx == npackets-1 ? (nelem/2) - idx*PKTSIZE : PKTSIZE;
        if (mype != root)
            nvshmem_signal_wait_until(mysync, NVSHMEM_CMP_GT, idx);
        nvshmem_put16((void*)pos, (void*)pos, elemcount, peer);
        nvshmem_fence();
        nvshmemx_signal_op(mysync, idx+1, NVSHMEM_SIGNAL_SET, peer);
    }
    
}

#undef CUDA_CHECK
#define CUDA_CHECK(stmt)                                                          \
    do {                                                                          \
        hipError_t result = (stmt);                                              \
        if (hipSuccess != result) {                                              \
            fprintf(stderr, "[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, \
                    hipGetErrorString(result));                                  \
            exit(-1);                                                             \
        }                                                                         \
    } while (0)

#define MPI_CHECK(stmt)                                                                         \
    do {                                                                                        \
        int result = (stmt);                                                                    \
        if (MPI_SUCCESS != result) {                                                            \
            fprintf(stderr, "[%s:%d] MPI failed with error %d \n", __FILE__, __LINE__, result); \
            exit(-1);                                                                           \
        }                                                                                       \
    } while (0)

__global__ void simple_shift(int *target, int mype, int npes) {
    int peer = (mype + 1) % npes;
    nvshmem_int_p(target, mype, peer);
}

int main(int c, char *v[]) {
    
    int rank, nranks;
    size_t data_len = NELEM;
    MPI_Comm mpi_comm;
    nvshmemx_init_attr_t attr;
    int mype, npes, mype_node;
    hipStream_t stream;

    MPI_CHECK(MPI_Init(&c, &v));
    MPI_CHECK(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CHECK(MPI_Comm_size(MPI_COMM_WORLD, &nranks));

    mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);

    // application picks the device each PE will use
    CUDA_CHECK(hipSetDevice(mype_node));
    CUDA_CHECK(hipStreamCreate(&stream));
    half *data = (half *)nvshmem_malloc(sizeof(half) * NELEM);
    half *data_h = (half *)malloc(sizeof(half) * NELEM);
    uint64_t *psync = (uint64_t *)nvshmem_calloc(2, sizeof(uint64_t));
    for (int i = 0; i < NELEM; i++) data_h[i] = (half)(mype+i);

    hipMemcpyAsync(data, data_h, sizeof(half) * NELEM, hipMemcpyHostToDevice, stream);
    int root = 0;
    dim3 gridDim(2), blockDim(1);
    void *args[] = {&data, &data_len, &root, &psync};

    nvshmemx_barrier_all_on_stream(stream);
    nvshmemx_collective_launch((const void *)ring_bcast, gridDim, blockDim, args, 0, stream);
    nvshmemx_barrier_all_on_stream(stream);

    hipMemcpyAsync(data_h, data, sizeof(half) * NELEM, hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    for (size_t i = 0; i < data_len; i++) {
        if ((int)data_h[i] != (int)i)
            printf("PE %d error, data[%zu] = %d expected data[%zu] = %d\n", mype, i, (int)data_h[i], i,
                   (int)i);
    }

    nvshmem_free(data);
    nvshmem_free(psync);
    free(data_h);

    nvshmem_finalize();
    MPI_CHECK(MPI_Finalize());
    return 0;
}

