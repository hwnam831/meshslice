#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <sstream>
#include <cstdlib>
#include "hip/hip_fp16.h"
// Utilities and system includes
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CUDA_RT_CALL(call)                                                                  \
    {                                                                                       \
        hipError_t cudaStatus = call;                                                      \
        if (hipSuccess != cudaStatus) {                                                    \
            fprintf(stderr,                                                                 \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                    "with "                                                                 \
                    "%s (%d).\n",                                                           \
                    #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
            exit( cudaStatus );                                                             \
        }                                                                                   \
    }

half* initMatrix(size_t H, size_t W, int val){
    //half* buf_h = (half *)malloc(sizeof(half) * H * W);
    //for (int i=0; i<H*W; i++)
    //    buf_h[i] = val;
    half* buf_d = NULL;
    CUDA_RT_CALL(hipMalloc(reinterpret_cast<void **>(&buf_d),
                             sizeof(half) * H * W));
    //CUDA_RT_CALL(hipMemcpy(buf_d, buf_h, sizeof(half) * H * W,
    //                         hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemset(buf_d, val, sizeof(half) * H * W));
    return buf_d;
}

#define M 1280
#define N 512
#define K 1024

int main(){

    half* A = initMatrix(M,K,1);
    half* B = initMatrix(M,K,2);
    half* C = initMatrix(M,N,0);
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasHandle_t handle;
    hipEvent_t start, stop;

    CUDA_RT_CALL(hipblasCreate(&handle));
    // Perform warmup operation with cublas
    CUDA_RT_CALL(hipblasHgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, MA,
        K, &alpha, B, N, A, K,
        &beta, C, N));
    half* C_h = (half *)malloc(sizeof(half) * M * N);
    CUDA_RT_CALL(hipMemcpy(C_h, C, sizeof(half) * M * N, hipMemcpyDeviceToHost));
    printf("C_h element %f\n", C_h[0]);
    free(C_h);
    CUDA_RT_CALL(hipblasDestroy(handle));
    CUDA_RT_CALL(hipFree(A));
    CUDA_RT_CALL(hipFree(B));
    return 0;
}

