#include <mpi.h>
//#include <nvshmem.h>
//#include <nvshmemx.h>

#include <cstdio>
#include <iostream>

#include <cstdlib>
#include "hip/hip_fp16.h"
// Utilities and system includes
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "cublas_helper.h"

#define M 2560
#define N 2048
#define K 3072
#define REPEAT 10

int main(){

    half* A = initMatrix(M,K,1);
    half* B = initMatrix(N,K,2);
    half* C = initMatrix(M,N,0);
    const half alpha = 1.0f;
    const half beta = 0.0f;
    hipblasHandle_t handle;
    hipEvent_t start, stop;
    
    checkCudaErrors(hipblasCreate(&handle));
    // Perform warmup operation with cublas
    hipDeviceSynchronize();
    checkCudaErrors(hipblasHgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M,
        K, &alpha, B, N, A, K,
        &beta, C, N));
    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    CUDA_RT_CALL(hipEventCreate(&start));
    CUDA_RT_CALL(hipEventCreate(&stop));

    // Record the start event
    CUDA_RT_CALL(hipEventRecord(start, NULL));

    for (int j = 0; j < REPEAT; j++) {
      // note cublas is column primary!
      // need to transpose the order
      checkCudaErrors(hipblasHgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M,
        K, &alpha, B, N, A, K,
        &beta, C, N));
    }

    printf("done.\n");

    // Record the stop event
    CUDA_RT_CALL(hipEventRecord(stop, NULL));

    // Wait for the stop event to complete
    CUDA_RT_CALL(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    CUDA_RT_CALL(hipEventElapsedTime(&msecTotal, start, stop));

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / REPEAT;
    double flopsPerMatrixMul = 2.0 * (double)M *
                               (double)N *
                               (double)K;
    double gigaFlops =
        (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
           gigaFlops, msecPerMatrixMul, flopsPerMatrixMul);

    half* C_h = (half *)malloc(sizeof(half) * M * N);
    CUDA_RT_CALL(hipMemcpy(C_h, C, sizeof(half) * M * N, hipMemcpyDeviceToHost));

    hipDeviceSynchronize();
    printf("C_h element %f\n", (float)C_h[0]);

    free(C_h);

    checkCudaErrors(hipblasDestroy(handle));
    CUDA_RT_CALL(hipFree(A));
    CUDA_RT_CALL(hipFree(B));
    return 0;
}

